#include "hip/hip_runtime.h"
// -*-c-*-
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);



__global__ void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < max) c[i] = a[i] + b[i];

}

void vectorAddCPU(int n, int* a, int*b, int*c) {
  for(int i=0; i<n; ++i) {
    c[i] = a[i] + b[i];
  }
}


int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c       
	int errors=0;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
	// Number of blocks must be enough for all N values without truncation
	dim3 blocksPerGrid((unsigned int)ceil(N / (double)THREADS_PER_BLOCK), 1, 1);
	dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);
	vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_a, d_b, d_c, N);
	checkCUDAError("CUDA kernel");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");

	// perform cpu version
	vectorAddCPU(N, a, b, c_ref);

	// check everything
	for(int i=0; i<N; ++i) {
	  if( (c[i] - c_ref[i])!=0) {
	    std::cout << "Error: mismatch for element " << i << " cpu = " << c_ref[i] << " gpu = " << c[i] << std::endl;
	    ++errors;
	  }
	}
	std::cout << "N(errors) = " << errors << std::endl;

	// Cleanup
	free(a); free(b); free(c); free(c_ref);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
